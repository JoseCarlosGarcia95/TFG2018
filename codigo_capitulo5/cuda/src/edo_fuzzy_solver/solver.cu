#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#include "hip/hip_runtime.h"
#include ""

#include "../../include/edo_fuzzy_solver/solver.cuh"
#include "../../include/utils/cuda.cuh"

__device__ double edo_fuzzy_solver_device_function_call(double x, double y) {
    return exp(-y * y);
}

__device__ unsigned long edo_fuzzy_solver_calculate_pointer(long i, long j, long n) {
    return i * (n+1) + j;
}

edo_fuzzy_solver_error edo_fuzzy_solver_create(double * fuzzy_set, unsigned int fuzzy_set_length, double tol, edo_fuzzy_solver* solver) {
    edo_fuzzy_solver_error error_t;

    error_t                  = EFS_E_OK;

    solver->fuzzy_set        = fuzzy_set;
    solver->fuzzy_set_length = fuzzy_set_length;
    solver->tol              = tol;

    return error_t;
}


__global__ void edo_fuzzy_solver_kernel_runge_kutta(double * fuzzy_set, double tol, unsigned int n, unsigned int fuzzy_set_length, double* x_values, double* y_values) {
    unsigned long i, j, pointer;
    double k1, k2, k3, k4, x, y;
    
    i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i >= fuzzy_set_length) {
        return;
    }

    // Initial value.
    
    pointer           = edo_fuzzy_solver_calculate_pointer(i, 0, fuzzy_set_length);
    y_values[pointer] = fuzzy_set[i];

    for (j = 0; j < n; j++) {
        pointer           = edo_fuzzy_solver_calculate_pointer(i, j, fuzzy_set_length);

        x_values[pointer] = j * tol;

        x                 = x_values[pointer];
        y                 = y_values[pointer];
        k1                = edo_fuzzy_solver_device_function_call(x, y);

        x                 = x_values[pointer] + .5 * tol;
        y                 = y_values[pointer] + .5 * tol * k1;
        k2                = edo_fuzzy_solver_device_function_call(x, y);

        x                 = x_values[pointer] + .5 * tol;
        y                 = y_values[pointer] + .5 * tol * k2;
        k3                = edo_fuzzy_solver_device_function_call(x, y);

        x                 = x_values[pointer] + tol;
        y                 = y_values[pointer] + tol * k3;
        k4                = edo_fuzzy_solver_device_function_call(x, y);

        y_values[pointer + 1] = y_values[pointer] + tol/6 * (k1 + 2*k2 + 2*k3 + k4);
    }
}

edo_fuzzy_solver_error edo_fuzzy_solver_solve(edo_fuzzy_solver solver, edo_fuzzy_solution * solution) {
    int gridSize, blockSize, minGridSize;
    unsigned long n, i, j;
    edo_fuzzy_solver_error error_t;
    hipError_t cuda_error;
    double* fuzzy_set_cuda, *x_values_cuda, *y_values_cuda, *x_values_raw, *y_values_raw;
    
    error_t = EFS_E_OK;
    n = 1.0/solver.tol;

    // Starts cuda.
    cuda_error = hipSetDevice(0);

    if (cuda_error != hipSuccess) {
        error_t = EFS_E_CUDA_ERROR;
        goto clean;
    }

    // Allocate cuda memory.
    cuda_error = hipMalloc(&fuzzy_set_cuda, sizeof(double) * solver.fuzzy_set_length);
    if (cuda_error != hipSuccess) {
        error_t = EFS_E_CUDA_ERROR;
        goto clean;
    }

    // Copy host memory to GPU.
    cuda_error = hipMemcpy(fuzzy_set_cuda, solver.fuzzy_set, solver.fuzzy_set_length * sizeof(double), hipMemcpyHostToDevice);
    if (cuda_error != hipSuccess) {
        error_t = EFS_E_CUDA_ERROR;
        goto clean;
    }

    cuda_error = hipMalloc(&x_values_cuda, sizeof(double) * solver.fuzzy_set_length * n);
    if (cuda_error != hipSuccess) {
        error_t = EFS_E_CUDA_ERROR;
        goto clean;
    }

    cuda_error = hipMalloc(&y_values_cuda, sizeof(double) * solver.fuzzy_set_length * n);
    if (cuda_error != hipSuccess) {
        error_t = EFS_E_CUDA_ERROR;
        goto clean;
    }

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, edo_fuzzy_solver_kernel_runge_kutta, 0, solver.fuzzy_set_length);
    gridSize = (solver.fuzzy_set_length + blockSize - 1) / blockSize;

    // Run runge kutta in kernel.    
    edo_fuzzy_solver_kernel_runge_kutta << <gridSize, blockSize >> >(fuzzy_set_cuda, solver.tol, n, solver.fuzzy_set_length, x_values_cuda, y_values_cuda);

    cuda_error = hipDeviceSynchronize();
    if (cuda_error != hipSuccess) {
        error_t = EFS_E_CUDA_ERROR;
        goto clean;
    }

    // Now pass the result to CPU.
    x_values_raw = (double*)malloc(sizeof(double) * solver.fuzzy_set_length * n);
    y_values_raw = (double*)malloc(sizeof(double) * solver.fuzzy_set_length * n);

    cuda_error = hipMemcpy(x_values_raw, x_values_cuda, solver.fuzzy_set_length * sizeof(double) * n, hipMemcpyDeviceToHost);
    if (cuda_error != hipSuccess) {
        error_t = EFS_E_CUDA_ERROR;
        goto clean;
    }

    cuda_error = hipMemcpy(y_values_raw, y_values_cuda, solver.fuzzy_set_length * sizeof(double) * n, hipMemcpyDeviceToHost);
    if (cuda_error != hipSuccess) {
        error_t = EFS_E_CUDA_ERROR;
        goto clean;
    }

    solution->x_values = (double**)realloc(solution->x_values, sizeof(double*) * solver.fuzzy_set_length);
    solution->y_values = (double**)realloc(solution->y_values, sizeof(double*) * solver.fuzzy_set_length);
    
    for (i = 0; i < solver.fuzzy_set_length;i++) {
        solution->x_values[i] = (double*)malloc(sizeof(double) * n);
        solution->y_values[i] = (double*)malloc(sizeof(double) * n);

        for (j = 0; j < n; j++) {
            solution->x_values[i][j] = x_values_raw[i * (solver.fuzzy_set_length + 1) + j];
            solution->y_values[i][j] = y_values_raw[i * (solver.fuzzy_set_length + 1) + j];
        }
    }

    solution->points = n;
 clean:
    if (!fuzzy_set_cuda) {
        hipFree(fuzzy_set_cuda);
    }
    
    if (!x_values_cuda) {
        hipFree(x_values_cuda);
    }

    if (!x_values_cuda) {
        hipFree(x_values_cuda);
    }

    if (!x_values_raw) {
        free(x_values_raw);
    }

    if (!y_values_raw) {
        free(y_values_raw);
    }
    
    if (cuda_error != hipSuccess) {
        fprintf(stderr,"CUDA assert: %s\n", hipGetErrorString(cuda_error));
    }
    return error_t;
}

double edo_fuzzy_solver_evaluate(double x, unsigned int fuzzy_value, edo_fuzzy_solution* solution) {
    return NAN; 
}
